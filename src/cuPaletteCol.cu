#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <ClqPart/cuPaletteCol.cuh>
#include <cub/device/device_scan.cuh>


__device__ __inline__ bool compare_pauli_matrices(
        const uint32_t * __restrict__ pauli1,
        const uint32_t * __restrict__ pauli2,
        const int pauliSize){
    uint32_t cnt = 0;
    for(int i = 0; i < pauliSize; i++){
        cnt += __popc(pauli1[i] & pauli2[i]);
    }
    if (cnt & 0x1) {
      return true;
    }
    else {
      return false;
    }
 }

__device__ __inline__ bool findFirstCommonElement(
        const NODE_T * __restrict__ colList1,
        const NODE_T * __restrict__ colList2,
        const NODE_T colSize) {
    int i = 0; // Index for colList1
    int j = 0; // Index for colList2

    while (i < colSize && j < colSize) {
        if (colList1[i] < colList2[j]) {
            i++; // Move to the next element in colList1
        } else if (colList1[i] > colList2[j]) {
            j++; // Move to the next element in colList2
        } else {
            return true; // Found a common element
        }
    }

    return false; // No common element found
}


// extern __shared__ uint32_t shared[];
__global__ void build_conflict_graph_kernel(
        const uint32_t *__restrict__ d_pauliEnc,
        const int pauliEncSize,
        const NODE_T *__restrict__ d_colList, 
        const NODE_T n_vertices, 
        const NODE_T n_colors,
        NODE_T *__restrict__ d_confOffsets, 
        NODE_T *__restrict__ d_confAdjList, 
        NODE_T *__restrict__ d_nConflicts){
    // NODE_T *s_pauliEnc = (uint32_t *)shared;
    // NODE_T *s_colList = (NODE_T *)&s_pauliEnc[pauliEncSize * shared_edges_size];
    int num_edges = n_vertices*n_vertices;
    // int block_edges = shared_edges_size * shared_edges_size;
    // Grid-Stride Loop
    for(int i = blockIdx.x * blockDim.x + threadIdx.x; i < num_edges; i += blockDim.x * gridDim.x){
        int row = i / n_vertices;
        int col = i % n_vertices;
        if(row != col){
            const uint32_t *pauli1 = &d_pauliEnc[row * pauliEncSize];
            const uint32_t *pauli2 = &d_pauliEnc[col * pauliEncSize];
            bool isedge = compare_pauli_matrices(pauli1, pauli2, pauliEncSize);
            // If conflicting complement edge
            if(!isedge){
                const NODE_T *colList1 = &d_colList[row * n_colors];
                const NODE_T *colList2 = &d_colList[col * n_colors];
                bool common_color = findFirstCommonElement(colList1, colList2, n_colors);
                if(common_color){
                    atomicAdd(d_nConflicts, 1);
                    int index_offset = atomicAdd(&d_confOffsets[row], 1);
                    d_confAdjList[row * n_vertices + index_offset] = col;
                }
            }
        }
    }
}

__global__ void build_complement_graph_kernel(
        const uint32_t *__restrict__ d_pauliEnc,
        const int pauliEncSize,
        const NODE_T *__restrict__ d_colList, 
        const NODE_T n_vertices, 
        const NODE_T n_colors,
        NODE_T *__restrict__ d_confOffsets, 
        NODE_T *__restrict__ d_confAdjList, 
        NODE_T *__restrict__ d_nConflicts){
    // NODE_T *s_pauliEnc = (uint32_t *)shared;
    // NODE_T *s_colList = (NODE_T *)&s_pauliEnc[pauliEncSize * shared_edges_size];
    int num_edges = n_vertices*n_vertices;
    // int block_edges = shared_edges_size * shared_edges_size;
    // Grid-Stride Loop
    for(int i = blockIdx.x * blockDim.x + threadIdx.x; i < num_edges; i += blockDim.x * gridDim.x){
        int row = i / n_vertices;
        int col = i % n_vertices;
        if(row != col){
            const uint32_t *pauli1 = &d_pauliEnc[row * pauliEncSize];
            const uint32_t *pauli2 = &d_pauliEnc[col * pauliEncSize];
            bool isedge = compare_pauli_matrices(pauli1, pauli2, pauliEncSize);
            // If conflicting complement edge
            if(!isedge){
                atomicAdd(d_nConflicts, 1);
                int index_offset = atomicAdd(&d_confOffsets[row], 1);
                d_confAdjList[row * n_vertices + index_offset] = col;
            }
        }
    }
}

template <typename OffsetTy>
__global__ void build_coo_complement_graph_kernel(
        const uint32_t *__restrict__ d_pauliEnc,
        const int pauliEncSize,
        const NODE_T *__restrict__ d_colList, 
        const NODE_T n_vertices, 
        const NODE_T n_colors,
        OffsetTy *__restrict__ d_confOffsets,  
        OffsetTy *__restrict__ d_nConflicts){
    // int num_edges = n_vertices*(n_vertices - 1)/2;
    OffsetTy num_edges = (OffsetTy)n_vertices*(OffsetTy)n_vertices;
    // NODE_T halfway_point = n_vertices / 2;
    // Grid-Stride Loop only for lower triangle of matrix
    OffsetTy conflict_count = 0;
    for(OffsetTy edge_id = blockIdx.x * blockDim.x + threadIdx.x; edge_id < num_edges; edge_id += blockDim.x * gridDim.x){
        NODE_T row = edge_id / (OffsetTy)n_vertices;
        NODE_T col = edge_id - ((OffsetTy)row * (OffsetTy)n_vertices);
        if(row > col){
            const uint32_t *pauli1 = &d_pauliEnc[row * pauliEncSize];
            const uint32_t *pauli2 = &d_pauliEnc[col * pauliEncSize];
            bool isedge = compare_pauli_matrices(pauli1, pauli2, pauliEncSize);
            // If conflicting complement edge
            if(!isedge){
                conflict_count++;
                atomicAdd(&d_confOffsets[row], 1);
                atomicAdd(&d_confOffsets[col], 1);
            }
        }
    }
    atomicAdd(d_nConflicts, conflict_count);
}

template <typename OffsetTy>
__global__ void build_coo_conflict_graph_kernel(
        const uint32_t *__restrict__ d_pauliEnc,
        const int pauliEncSize,
        const NODE_T *__restrict__ d_colList, 
        const NODE_T n_vertices, 
        const NODE_T n_colors,
        OffsetTy *__restrict__ d_confOffsets, 
        NODE_T *__restrict__ d_confAdjList, 
        OffsetTy *__restrict__ d_nConflicts){
    Edge *d_cooEdgeList = (Edge *)d_confAdjList;
    // int num_edges = n_vertices*(n_vertices - 1)/2;
    OffsetTy num_edges = (OffsetTy)n_vertices*(OffsetTy)n_vertices;
    // NODE_T halfway_point = n_vertices / 2;
    // Grid-Stride Loop only for lower triangle of matrix
    for(OffsetTy edge_id = blockIdx.x * blockDim.x + threadIdx.x; edge_id < num_edges; edge_id += blockDim.x * gridDim.x){
        NODE_T row = edge_id / (OffsetTy)n_vertices;
        NODE_T col = edge_id - ((OffsetTy)row * (OffsetTy)n_vertices);
        // Equivalent to edge_id % n_vertices
        if(row > col){
            const uint32_t *pauli1 = &d_pauliEnc[row * pauliEncSize];
            const uint32_t *pauli2 = &d_pauliEnc[col * pauliEncSize];
            bool isedge = compare_pauli_matrices(pauli1, pauli2, pauliEncSize);
            // If conflicting complement edge
            if(!isedge){
                const NODE_T *colList1 = &d_colList[row * n_colors];
                const NODE_T *colList2 = &d_colList[col * n_colors];
                bool common_color = findFirstCommonElement(colList1, colList2, n_colors);
                if(common_color){
                    OffsetTy index_offset = atomicAdd(d_nConflicts, 1);
                    atomicAdd(&d_confOffsets[row], 1);
                    atomicAdd(&d_confOffsets[col], 1);
                    d_cooEdgeList[index_offset] = Edge{row, col};
                }
            }
        }
    }
}

template <typename OffsetTy>
__global__ void build_coo_conflict_graph_kernel(
        const uint32_t *__restrict__ d_pauliEnc,
        const int pauliEncSize,
        const NODE_T *__restrict__ d_colList, 
        const NODE_T *__restrict__ d_nodeList,
        const NODE_T n_vertices, 
        const NODE_T n_colors,
        OffsetTy *__restrict__ d_confOffsets, 
        NODE_T *__restrict__ d_confAdjList, 
        OffsetTy *__restrict__ d_nConflicts){
    Edge *d_cooEdgeList = (Edge *)d_confAdjList;
    // int num_edges = n_vertices*(n_vertices - 1)/2;
    OffsetTy num_edges = (OffsetTy)n_vertices*(OffsetTy)n_vertices;
    // NODE_T halfway_point = n_vertices / 2;
    // Grid-Stride Loop only for lower triangle of matrix
    for(OffsetTy edge_id = blockIdx.x * blockDim.x + threadIdx.x; edge_id < num_edges; edge_id += blockDim.x * gridDim.x){
        NODE_T row = edge_id / (OffsetTy)n_vertices;
        NODE_T col = edge_id - ((OffsetTy)row * (OffsetTy)n_vertices);
        // Equivalent to edge_id % n_vertices
        if(row > col){
            const NODE_T row_mapped = d_nodeList[row];
            const NODE_T col_mapped = d_nodeList[col];
            const uint32_t *pauli1 = &d_pauliEnc[row_mapped * pauliEncSize];
            const uint32_t *pauli2 = &d_pauliEnc[col_mapped * pauliEncSize];
            bool isedge = compare_pauli_matrices(pauli1, pauli2, pauliEncSize);
            // If conflicting complement edge
            if(!isedge){
                const NODE_T *colList1 = &d_colList[row * n_colors];
                const NODE_T *colList2 = &d_colList[col * n_colors];
                bool common_color = findFirstCommonElement(colList1, colList2, n_colors);
                if(common_color){
                    OffsetTy index_offset = atomicAdd(d_nConflicts, 1);
                    atomicAdd(&d_confOffsets[row_mapped], 1);
                    atomicAdd(&d_confOffsets[col_mapped], 1);
                    d_cooEdgeList[index_offset] = Edge{row_mapped, col_mapped};
                }
            }
        }
    }
}

template <typename OffsetTy>
__global__ void build_csr_conflict_graph_kernel(
        const NODE_T n_vertices, 
        const OffsetTy num_conf_edges,
        const OffsetTy *__restrict__ d_confOffsets, 
        OffsetTy *__restrict__ d_confOffsetsCnt, 
        const NODE_T *__restrict__ d_confAdjList,
        NODE_T *__restrict__ d_confCsr){
    Edge *d_cooEdgeList = (Edge *)d_confAdjList;
    for(OffsetTy edge_id = blockIdx.x * blockDim.x + threadIdx.x; edge_id < num_conf_edges; edge_id += blockDim.x * gridDim.x){
        Edge edge = d_cooEdgeList[edge_id];
        NODE_T u = edge.u;
        NODE_T v = edge.v;
        OffsetTy u_offset = atomicAdd(&d_confOffsetsCnt[u], 1);
        d_confCsr[d_confOffsets[u] + u_offset] = v;
        OffsetTy v_offset = atomicAdd(&d_confOffsetsCnt[v], 1);
        d_confCsr[d_confOffsets[v] + v_offset] = u;
    }
}

void buildCompGraphDevice(
        const uint32_t *d_pauliEnc,
        const int pauliEncSize,
        const NODE_T *d_colList,
        const NODE_T n_vertices,
        const NODE_T n_colors,
        NODE_T *d_confOffsets,
        NODE_T *d_confAdjList,
        NODE_T *d_nConflicts){
    // Find cuda properties
    int device;
    hipDeviceProp_t prop;
    hipGetDevice(&device);
    hipGetDeviceProperties(&prop, device);
    int nSM = prop.multiProcessorCount;
    int maxThreadsPerSM = prop.maxThreadsPerMultiProcessor;
    int block_size = 256;
    int num_blocks = nSM * (maxThreadsPerSM / block_size);
    build_complement_graph_kernel<<<num_blocks, block_size>>>(d_pauliEnc, pauliEncSize, d_colList, n_vertices, n_colors, d_confOffsets, d_confAdjList, d_nConflicts);
}

void buildConfGraphDevice(
        const uint32_t *d_pauliEnc,
        const int pauliEncSize,
        const NODE_T *d_colList,
        const NODE_T n_vertices,
        const NODE_T n_colors,
        NODE_T *d_confOffsets,
        NODE_T *d_confAdjList,
        NODE_T *d_nConflicts){
    // Find cuda properties
    int device;
    hipDeviceProp_t prop;
    hipGetDevice(&device);
    hipGetDeviceProperties(&prop, device);
    int nSM = prop.multiProcessorCount;
    int maxThreadsPerSM = prop.maxThreadsPerMultiProcessor;
    int block_size = 256;
    int num_blocks = nSM * (maxThreadsPerSM / block_size);
    build_conflict_graph_kernel<<<num_blocks, block_size>>>(d_pauliEnc, pauliEncSize, d_colList, n_vertices, n_colors, d_confOffsets, d_confAdjList, d_nConflicts);
}

template <typename OffsetTy>
void buildCooConfGraphDevice(
        const uint32_t *d_pauliEnc,
        const int pauliEncSize,
        const NODE_T *d_colList,
        const NODE_T n_vertices,
        const NODE_T n_colors,
        OffsetTy *d_confOffsets,
        NODE_T *d_confAdjList,
        OffsetTy *d_nConflicts){
    // Find cuda properties
    int device;
    hipDeviceProp_t prop;
    hipGetDevice(&device);
    hipGetDeviceProperties(&prop, device);
    int nSM = prop.multiProcessorCount;
    int maxThreadsPerSM = prop.maxThreadsPerMultiProcessor;
    int block_size = 256;
    int num_blocks = nSM * (maxThreadsPerSM / block_size);
    build_coo_conflict_graph_kernel<<<num_blocks, block_size>>>(d_pauliEnc, pauliEncSize, d_colList, n_vertices, n_colors, d_confOffsets, d_confAdjList, d_nConflicts);
}

template <typename OffsetTy>
void buildCooConfGraphDevice(
        const uint32_t *d_pauliEnc,
        const int pauliEncSize,
        const NODE_T *d_colList,
        const NODE_T *d_nodeList,
        const NODE_T n_vertices,
        const NODE_T n_colors,
        OffsetTy *d_confOffsets,
        NODE_T *d_confAdjList,
        OffsetTy *d_nConflicts){
    // Find cuda properties
    int device;
    hipDeviceProp_t prop;
    hipGetDevice(&device);
    hipGetDeviceProperties(&prop, device);
    int nSM = prop.multiProcessorCount;
    int maxThreadsPerSM = prop.maxThreadsPerMultiProcessor;
    int block_size = 256;
    int num_blocks = nSM * (maxThreadsPerSM / block_size);
    build_coo_conflict_graph_kernel<<<num_blocks, block_size>>>(d_pauliEnc, pauliEncSize, d_colList, d_nodeList, n_vertices, n_colors, d_confOffsets, d_confAdjList, d_nConflicts);
}

template <typename OffsetTy>
void buildCooCompGraphDevice(
        const uint32_t *d_pauliEnc,
        const int pauliEncSize,
        const NODE_T *d_colList,
        const NODE_T n_vertices,
        const NODE_T n_colors,
        OffsetTy *d_confOffsets,
        OffsetTy *d_nConflicts){
    // Find cuda properties
    int device;
    hipDeviceProp_t prop;
    hipGetDevice(&device);
    hipGetDeviceProperties(&prop, device);
    int nSM = prop.multiProcessorCount;
    int maxThreadsPerSM = prop.maxThreadsPerMultiProcessor;
    int block_size = 256;
    int num_blocks = nSM * (maxThreadsPerSM / block_size);
    build_coo_complement_graph_kernel<<<num_blocks, block_size>>>(d_pauliEnc, pauliEncSize, d_colList, n_vertices, n_colors, d_confOffsets, d_nConflicts);
}

template <typename OffsetTy>
void buildCsrConfGraphDevice(
        const NODE_T n_vertices,
        const OffsetTy *d_confOffsets,
        OffsetTy *d_confOffsetsCnt,
        const NODE_T *d_confAdjList,
        NODE_T *d_confCsr,
        const OffsetTy nConflicts){
    hipMemset(d_confOffsetsCnt, 0, n_vertices * sizeof(OffsetTy));
    hipDeviceSynchronize();
    // Call kernel to generate the CSR
    int device;
    hipDeviceProp_t prop;
    hipGetDevice(&device);
    hipGetDeviceProperties(&prop, device);
    int nSM = prop.multiProcessorCount;
    int maxThreadsPerSM = prop.maxThreadsPerMultiProcessor;
    int block_size = 256;
    int num_blocks = nSM * (maxThreadsPerSM / block_size);
    build_csr_conflict_graph_kernel<<<num_blocks, block_size>>>(n_vertices, nConflicts, d_confOffsets, d_confOffsetsCnt, d_confAdjList, d_confCsr);
}

template <typename OffsetTy>
__host__ void cubInclusiveSum(void *d_confOffsetsCnt, const NODE_T n, OffsetTy *d_confOffsets){
    int num_elements = n + 1;
    size_t num_bytes = num_elements * sizeof(OffsetTy);
    // Call cub to perform inclusive sum
    hipcub::DeviceScan::ExclusiveSum<OffsetTy *, OffsetTy *>(d_confOffsetsCnt, num_bytes, d_confOffsets, d_confOffsets, num_elements);
}

// Create forced instantiation of templates for NODE_T, unsigned int, and unsigned long long
template void buildCooConfGraphDevice(const unsigned int *, const int, const NODE_T *, const NODE_T, const NODE_T, unsigned int *, NODE_T *, unsigned int *);
template void buildCooConfGraphDevice(const unsigned int *, const int, const NODE_T *, const NODE_T, const NODE_T, unsigned long long *, NODE_T *, unsigned long long *);
template void buildCooConfGraphDevice(const unsigned int *, const int, const NODE_T *, const NODE_T, const NODE_T, NODE_T *, NODE_T *, NODE_T *);
template void buildCooConfGraphDevice(const unsigned int *, const int, const NODE_T *, const NODE_T *, const NODE_T, const NODE_T, unsigned int *, NODE_T *, unsigned int *);
template void buildCooConfGraphDevice(const unsigned int *, const int, const NODE_T *, const NODE_T *, const NODE_T, const NODE_T, unsigned long long *, NODE_T *, unsigned long long *);
template void buildCooConfGraphDevice(const unsigned int *, const int, const NODE_T *, const NODE_T *, const NODE_T, const NODE_T, NODE_T *, NODE_T *, NODE_T *);
template void buildCooCompGraphDevice(const unsigned int *, const int, const NODE_T *, const NODE_T, const NODE_T, unsigned int *, unsigned int *);
template void buildCooCompGraphDevice(const unsigned int *, const int, const NODE_T *, const NODE_T, const NODE_T, unsigned long long *, unsigned long long *);
template void buildCooCompGraphDevice(const unsigned int *, const int, const NODE_T *, const NODE_T, const NODE_T, NODE_T *, NODE_T *);
template void buildCsrConfGraphDevice(const NODE_T, const unsigned int *, unsigned int *, const NODE_T *, NODE_T *, const unsigned int);
template void buildCsrConfGraphDevice(const NODE_T, const unsigned long long *, unsigned long long *, const NODE_T *, NODE_T *, const unsigned long long);
template void buildCsrConfGraphDevice(const NODE_T, const NODE_T *, NODE_T *, const NODE_T *, NODE_T *, const NODE_T);
template void cubInclusiveSum(void *, const NODE_T, unsigned int *);
template void cubInclusiveSum(void *, const NODE_T, unsigned long long *);
template void cubInclusiveSum(void *, const NODE_T, NODE_T *);
