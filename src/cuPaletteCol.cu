#include "hip/hip_runtime.h"
#include "ClqPart/cuPaletteCol.cuh"
#include <hip/hip_runtime.h>

__device__ __inline__ bool compare_pauli_matrices(
        const uint32_t * __restrict__ pauli1,
        const uint32_t * __restrict__ pauli2,
        const int pauliSize){
    uint32_t cnt = 0;
    for(int i = 0; i < pauliSize; i++){
        cnt += __popc(pauli1[i] & pauli2[i]);
    }
    if (cnt & 0x1) {
      return true;
    }
    else {
      return false;
    }
 }

__device__ __inline__ bool findFirstCommonElement(
        const NODE_T * __restrict__ colList1,
        const NODE_T * __restrict__ colList2,
        const NODE_T colSize) {
    int i = 0; // Index for colList1
    int j = 0; // Index for colList2

    while (i < colSize && j < colSize) {
        if (colList1[i] < colList2[j]) {
            i++; // Move to the next element in colList1
        } else if (colList1[i] > colList2[j]) {
            j++; // Move to the next element in colList2
        } else {
            return true; // Found a common element
        }
    }

    return false; // No common element found
}


// extern __shared__ uint32_t shared[];
__global__ void build_conf_graph_kernel(
        const uint32_t *__restrict__ d_pauliEnc,
        const int pauliEncSize,
        const NODE_T *__restrict__ d_colList, 
        const NODE_T n_vertices, 
        const NODE_T n_colors,
        NODE_T *__restrict__ d_confOffsets, 
        NODE_T *__restrict__ d_confAdjList, 
        NODE_T *__restrict__ d_nConflicts){
    // NODE_T *s_pauliEnc = (uint32_t *)shared;
    // NODE_T *s_colList = (NODE_T *)&s_pauliEnc[pauliEncSize * shared_edges_size];
    int num_edges = n_vertices*n_vertices;
    // int block_edges = shared_edges_size * shared_edges_size;
    // Grid-Stride Loop
    for(int i = blockIdx.x * blockDim.x + threadIdx.x; i < num_edges; i += blockDim.x * gridDim.x){
        int row = i / n_vertices;
        int col = i % n_vertices;
        if(row != col){
            const uint32_t *pauli1 = &d_pauliEnc[row * pauliEncSize];
            const uint32_t *pauli2 = &d_pauliEnc[col * pauliEncSize];
            bool isedge = compare_pauli_matrices(pauli1, pauli2, pauliEncSize);
            // If conflicting complement edge
            if(!isedge){
                const NODE_T *colList1 = &d_colList[row * n_colors];
                const NODE_T *colList2 = &d_colList[col * n_colors];
                bool common_color = findFirstCommonElement(colList1, colList2, n_colors);
                if(common_color){
                    atomicAdd(d_nConflicts, 1);
                    int index_offset = atomicAdd(&d_confOffsets[row], 1);
                    d_confAdjList[row * n_vertices + index_offset] = col;
                }
            }
        }
    }
}

void buildConfGraphDevice(
        const uint32_t *d_pauliEnc,
        const int pauliEncSize,
        const NODE_T *d_colList,
        const NODE_T n_vertices,
        const NODE_T n_colors,
        NODE_T *d_confOffsets,
        NODE_T *d_confAdjList,
        NODE_T *d_nConflicts){
    // Find cuda properties
    int device;
    hipDeviceProp_t prop;
    hipGetDevice(&device);
    hipGetDeviceProperties(&prop, device);
    int nSM = prop.multiProcessorCount;
    int maxThreadsPerSM = prop.maxThreadsPerMultiProcessor;
    int block_size = 256;
    int num_blocks = nSM * (maxThreadsPerSM / block_size);
    build_conf_graph_kernel<<<num_blocks, block_size>>>(d_pauliEnc, pauliEncSize, d_colList, n_vertices, n_colors, d_confOffsets, d_confAdjList, d_nConflicts);
}